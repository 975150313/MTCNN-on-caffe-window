#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <cmath>

#include "caffe/layers/label_specific_hard_margin.hpp"

namespace caffe {

  template <typename Dtype>
  __global__ void LabelSpecificHardMarginForward(const int num, const int dim, const Dtype* bottom_data, const Dtype* label,
                                                 Dtype* row_sum, Dtype* top_data, Dtype positive_weight) {
    CUDA_KERNEL_LOOP(index, num) {
      int gt = static_cast<int>(label[index]);
      row_sum[index] = (row_sum[index] - bottom_data[index * dim + gt]) / (dim - 1);
      top_data[index * dim + gt] = bottom_data[index * dim + gt] * positive_weight + row_sum[index] * (1 - positive_weight);
      row_sum[index] = top_data[index * dim + gt] - bottom_data[index * dim + gt];
    }
  }

  template <typename Dtype>
  __global__ void LabelSpecificHardMarginBackward(const int num, const int dim, const Dtype* top_diff, const Dtype* label,
                                              const Dtype* bottom_data, Dtype* bottom_diff, Dtype positive_weight) {
    CUDA_KERNEL_LOOP(index, num) {
      int gt = static_cast<int>(label[index]);
      bottom_diff[index * dim + gt] = top_diff[index * dim + gt] * positive_weight;
    }
  }

  template <typename Dtype>
  __global__ void LabelSpecificHardMarginBackwardNegative(const int num, const int dim, const Dtype* top_diff, const Dtype* label,
    const Dtype* bottom_data, Dtype* bottom_diff, Dtype positive_weight) {
    CUDA_KERNEL_LOOP(index, num * dim) {
      int n = index / dim;
      int d = index % dim;
      int gt = static_cast<int>(label[n]);
      if (d != gt) {
        bottom_diff[n * dim + d] += top_diff[n * dim + gt] * (1 - positive_weight) / (dim - 1);
      }
    }
  }

  template <typename Dtype>
  void LabelSpecificHardMarginLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                                    const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* label_data = bottom[1]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();

    int num = bottom[0]->num();
    int count = bottom[0]->count();
    int dim = count / num;

    caffe_copy(count, bottom_data, top_data);
    if (this->phase_ == TEST) return;

    caffe_gpu_gemv(CblasNoTrans, num, dim, Dtype(1), bottom_data, sum_multiplier_.gpu_data(), Dtype(0), margins_.mutable_gpu_data());

    // NOLINT_NEXT_LINE(whitespace/operators)
    LabelSpecificHardMarginForward<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
      num, dim, bottom_data, label_data, margins_.mutable_gpu_data(), top_data, positive_weight);
    CUDA_POST_KERNEL_CHECK;

    if (top.size() == 2) {
      top[1]->mutable_cpu_data()[0] = margins_.asum_data() / Dtype(num) / Dtype(M_PI) * Dtype(180.0);
    }
  }

  template <typename Dtype>
  void LabelSpecificHardMarginLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                                     const vector<bool>& propagate_down,
                                                     const vector<Blob<Dtype>*>& bottom) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* label_data = bottom[1]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* top_data = top[0]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

    int num = bottom[0]->num();
    int count = bottom[0]->count();
    int dim = count / num;
    if (propagate_down[0]) {
      caffe_copy(count, top_diff, bottom_diff);
      if (this->phase_ == TEST) return;

      // NOLINT_NEXT_LINE(whitespace/operators)
      LabelSpecificHardMarginBackward<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
        num, dim, top_diff, label_data, bottom_data, bottom_diff, positive_weight);
      CUDA_POST_KERNEL_CHECK;

      // NOLINT_NEXT_LINE(whitespace/operators)
      LabelSpecificHardMarginBackwardNegative<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> > (
        num, dim, top_diff, label_data, bottom_data, bottom_diff, positive_weight);
      CUDA_POST_KERNEL_CHECK;
    }
  }


  INSTANTIATE_LAYER_GPU_FUNCS(LabelSpecificHardMarginLayer);


}  // namespace caffe
